//****************************************************************
// Optimized GPU MD5 brute-forcer – annotated version (dynamic PASSWORD_LEN)
//****************************************************************
#include <cstdio>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <chrono>

// 1) Configurable charset and length as macros
#define HOST_CHARSET \
    "abcdefghijklmnopqrstuvwxyzABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789"
#define HOST_CHARSET_SIZE (sizeof(HOST_CHARSET)-1)
#define PASSWORD_LEN      6

/* ---------- early-exit globals (managed) ---------- */
__device__ __managed__ volatile int      g_found = 0;
__device__ __managed__ uint64_t          g_idx   = 0;

/* ---------- constant memory ---------- */
__device__ __constant__ uint32_t d_K[64];
__device__ __constant__ int d_r[64] = {
   7,12,17,22,7,12,17,22,7,12,17,22,7,12,17,22,
   5, 9,14,20,5, 9,14,20,5, 9,14,20,5, 9,14,20,
   4,11,16,23,4,11,16,23,4,11,16,23,4,11,16,23,
   6,10,15,21,6,10,15,21,6,10,15,21,6,10,15,21
};
__device__ __constant__ char d_CHARSET[HOST_CHARSET_SIZE+1] = HOST_CHARSET;

/* ---------- host-to-device constant upload ---------- */
void upload_constants() {
    static const uint32_t h_K[64] = {
        0xd76aa478,0xe8c7b756,0x242070db,0xc1bdceee,0xf57c0faf,0x4787c62a,0xa8304613,0xfd469501,
        0x698098d8,0x8b44f7af,0xffff5bb1,0x895cd7be,0x6b901122,0xfd987193,0xa679438e,0x49b40821,
        0xf61e2562,0xc040b340,0x265e5a51,0xe9b6c7aa,0xd62f105d,0x02441453,0xd8a1e681,0xe7d3fbc8,
        0x21e1cde6,0xc33707d6,0xf4d50d87,0x455a14ed,0xa9e3e905,0xfcefa3f8,0x676f02d9,0x8d2a4c8a,
        0xfffa3942,0x8771f681,0x6d9d6122,0xfde5380c,0xa4beea44,0x4bdecfa9,0xf6bb4b60,0xbebfbc70,
        0x289b7ec6,0xeaa127fa,0xd4ef3085,0x04881d05,0xd9d4d039,0xe6db99e5,0x1fa27cf8,0xc4ac5665,
        0xf4292244,0x432aff97,0xab9423a7,0xfc93a039,0x655b59c3,0x8f0ccc92,0xffeff47d,0x85845dd1,
        0x6fa87e4f,0xfe2ce6e0,0xa3014314,0x4e0811a1,0xf7537e82,0xbd3af235,0x2ad7d2bb,0xeb86d391
    };
    hipMemcpyToSymbol(HIP_SYMBOL(d_K), h_K, sizeof(h_K));
}

/* ---------- utility functions ---------- */
__device__ __forceinline__ uint32_t leftrotate(uint32_t x, uint32_t c) {
    return (x << c) | (x >> (32 - c));
}

__host__ unsigned char hex2byte(char hi, char lo) {
    auto val = [&](char c)->int {
        if (c >= '0' && c <= '9') return c - '0';
        if (c >= 'a' && c <= 'f') return c - 'a' + 10;
        if (c >= 'A' && c <= 'F') return c - 'A' + 10;
        return 0;
    };
    return (val(hi) << 4) | val(lo);
}

/* ---------- single-block MD5 (dynamic-length version) ---------- */
__device__ __forceinline__ void md5_single(const char* in, unsigned char dig[16]) {
    uint32_t M[16];
    // zero-out block
    #pragma unroll
    for (int i = 0; i < 16; ++i) M[i] = 0;

    // copy PASSWORD_LEN bytes
    #pragma unroll
    for (int i = 0; i < PASSWORD_LEN; ++i) {
        int word = i >> 2;
        int shift = (i & 3) * 8;
        M[word] |= ((uint32_t)in[i] << shift);
    }
    // append 0x80 bit
    {
        int i = PASSWORD_LEN;
        int word = i >> 2;
        int shift = (i & 3) * 8;
        M[word] |= (0x80u << shift);
    }
    // message length in bits
    M[14] = (uint32_t)PASSWORD_LEN * 8;
    M[15] = 0;

    uint32_t a = 0x67452301, b = 0xefcdab89,
             c = 0x98badcfe, d = 0x10325476;

    #pragma unroll 64
    for (int i = 0; i < 64; ++i) {
        uint32_t F, g;
        if (i < 16)       { F = (b & c) | (~b & d);       g = i;      }
        else if (i < 32)  { F = (d & b) | (~d & c);       g = (5*i+1)&15; }
        else if (i < 48)  { F = b ^ c ^ d;                g = (3*i+5)&15; }
        else              { F = c ^ (b | ~d);            g = (7*i)&15;   }
        F += a + d_K[i] + M[g];
        a = d; d = c; c = b; b += leftrotate(F, d_r[i]);
    }
    a += 0x67452301; b += 0xefcdab89;
    c += 0x98badcfe; d += 0x10325476;

    uint32_t regs[4] = {a,b,c,d};
    #pragma unroll
    for (int i = 0; i < 4; ++i) {
        dig[4*i]   = regs[i] & 0xFF;
        dig[4*i+1] = (regs[i] >> 8) & 0xFF;
        dig[4*i+2] = (regs[i] >> 16) & 0xFF;
        dig[4*i+3] = (regs[i] >> 24) & 0xFF;
    }
}

/* ---------- brute-force kernel ---------- */
__global__ __launch_bounds__(256,4)
void brute7(const unsigned char* target_bin, uint64_t total) {
    if (g_found) return;
    uint64_t stride = blockDim.x * (uint64_t)gridDim.x;
    uint64_t idx    = blockIdx.x * blockDim.x + threadIdx.x;

    uint4 t4 = *((uint4*)target_bin);
    unsigned char dig[16];
    char pw[PASSWORD_LEN+1]; pw[PASSWORD_LEN] = '\0';

    for (; idx < total; idx += stride) {
        if (g_found) return;
        uint64_t v = idx;
        #pragma unroll
        for (int p = PASSWORD_LEN-1; p >= 0; --p) {
            pw[p] = d_CHARSET[v % HOST_CHARSET_SIZE];
            v /= HOST_CHARSET_SIZE;
        }
        md5_single(pw, dig);
        uint4 d4 = *((uint4*)dig);
        if (d4.x==t4.x && d4.y==t4.y && d4.z==t4.z && d4.w==t4.w) {
            if (atomicCAS((int*)&g_found,0,1)==0) g_idx = idx;
            return;
        }
    }
}

/* ---------- host driver ---------- */
int main(int argc, char** argv) {
    if (argc != 2) {
        printf("Usage: %s <32-char MD5 hex>\n", argv[0]);
        return 1;
    }
    upload_constants();

    unsigned char h_target[16];
    for (int i = 0; i < 16; ++i) {
        h_target[i] = hex2byte(argv[1][2*i], argv[1][2*i+1]);
    }
    unsigned char* d_target;
    hipMalloc(&d_target, 16);
    hipMemcpy(d_target, h_target, 16, hipMemcpyHostToDevice);

    uint64_t total = 1;
    for (int i = 0; i < PASSWORD_LEN; ++i) total *= HOST_CHARSET_SIZE;

    const int blocks = 1024, threads = 256;
    auto t0 = std::chrono::steady_clock::now();
    brute7<<<blocks,threads>>>(d_target, total);
    hipDeviceSynchronize();
    double sec = std::chrono::duration<double>(
        std::chrono::steady_clock::now() - t0).count();

    int h_found = 0;
    uint64_t h_idx = 0;
    hipMemcpyFromSymbol(&h_found, HIP_SYMBOL(g_found), sizeof(int));
    hipMemcpyFromSymbol(&h_idx,   HIP_SYMBOL(g_idx),   sizeof(uint64_t));

    if (h_found) {
        char pw[PASSWORD_LEN+1];
        uint64_t v = h_idx;
        for (int p = PASSWORD_LEN-1; p >= 0; --p) {
            pw[p] = HOST_CHARSET[v % HOST_CHARSET_SIZE];
            v /= HOST_CHARSET_SIZE;
        }
        pw[PASSWORD_LEN] = '\0';
        printf("Password found : %s\n", pw);
    } else {
        puts("Password NOT found.");
    }
    printf("GPU elapsed     : %.6f s  (%.2f Ghash/s)\n", sec, total/sec/1e9);
    hipFree(d_target);
    return 0;
}
